#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <vector>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <cutf/memory.hpp>
#include <cutf/type.hpp>
#include <cutf/error.hpp>
#include "tsqr.hpp"
#include "tcqr.hpp"
#include "utils.hpp"
#include "matrix_copy.cuh"
#include "matrix_operations.cuh"
#include "gemm_core/gemm_core.cuh"

//#define DEBUG
//#define DEBUG_INPUT_MATRIX_PRINT
//#define DEBUG_Q_MATRIX_PRINT

namespace{
constexpr unsigned warp_size = 32;
template <class Func>
void debug_func(Func func) {
#ifdef DEBUG
	func();
#endif
}
std::size_t get_batch_size_log2(const std::size_t m) {
	return (std::max(5u, static_cast<unsigned>( std::ceil( std::log2(static_cast<float>(m))))) - 5u);
}
std::size_t get_batch_size(const std::size_t m) {
	return 1lu << get_batch_size_log2(m);
}

// backward 1層目以外
template <bool UseTC, class T>
__global__ void tsqr_backward(
		T* const ac_ptr,
		const T* const b_ptr,
		const unsigned n,
		const std::size_t k
		) {
	constexpr std::size_t FRAGMENT_DIM_M = 32;
	constexpr std::size_t FRAGMENT_DIM_N = 16;
	constexpr std::size_t max_batch_size_per_block = 4;
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto matrix_id = tid / warp_size;
	const auto shared_memory_id = matrix_id % max_batch_size_per_block;
	const auto ac_m = (1lu << (k)) * 2 * n;

	if(matrix_id >= (1lu << k)) return;

	__shared__ T shared_ac_in[FRAGMENT_DIM_M * FRAGMENT_DIM_N * max_batch_size_per_block];
	__shared__ T shared_ac_out[FRAGMENT_DIM_M * FRAGMENT_DIM_N * max_batch_size_per_block];
	__shared__ T shared_b[FRAGMENT_DIM_N * FRAGMENT_DIM_N * max_batch_size_per_block];

	const auto shared_ac_in_ptr = shared_ac_in + FRAGMENT_DIM_M * FRAGMENT_DIM_N * shared_memory_id;
	const auto shared_ac_out_ptr = shared_ac_out + FRAGMENT_DIM_M * FRAGMENT_DIM_N * shared_memory_id;
	const auto shared_b_ptr = shared_b + FRAGMENT_DIM_N * FRAGMENT_DIM_N * shared_memory_id;

	// AC(in)のコピー
	mtk::matrix_copy::g2s32x16_1w(
			shared_ac_in_ptr, 2 * n, n,
			ac_ptr, matrix_id * 2 * n, ac_m,
			tid
			);
	// Bのコピー
	mtk::matrix_copy::g2s16x16_1w(
			shared_b_ptr, n, n,
			b_ptr, matrix_id * n, ac_m / 2,
			tid
			);
	// AC(out)の初期化
	mtk::matrix_operation::make_zero_matrix<T, FRAGMENT_DIM_M, FRAGMENT_DIM_N, 1>(
			shared_ac_out_ptr, tid);

	__syncthreads();

	mtk::gemm_core16x16<T, 1>(
			shared_ac_out_ptr, FRAGMENT_DIM_M,
			shared_ac_in_ptr, FRAGMENT_DIM_M,
			shared_b_ptr, FRAGMENT_DIM_N,
			tid & 0x1f
			);

	mtk::gemm_core16x16<T, 1>(
			shared_ac_out_ptr + FRAGMENT_DIM_N, FRAGMENT_DIM_M,
			shared_ac_in_ptr + FRAGMENT_DIM_N, FRAGMENT_DIM_M,
			shared_b_ptr, FRAGMENT_DIM_N,
			tid & 0x1f
			);

	__syncthreads();

	mtk::matrix_copy::s2g32x16_1w(
			ac_ptr, matrix_id * 2 * n, ac_m,
			shared_ac_out_ptr, 2 * n, n,
			tid
			);
}
template <>
__global__ void tsqr_backward<true, half>(
		half* const ac_ptr,
		const half* const b_ptr,
		const unsigned n,
		const std::size_t k
		) {
	constexpr std::size_t FRAGMENT_DIM_M = 32;
	constexpr std::size_t FRAGMENT_DIM_N = 16;
	constexpr std::size_t max_batch_size_per_block = 4;
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto matrix_id = tid / warp_size;
	const auto shared_memory_id = matrix_id % max_batch_size_per_block;
	const auto ac_m = (1lu << (k)) * 2 * n;

	if(matrix_id >= (1lu << k)) return;

	__shared__ half shared_ac_f16[FRAGMENT_DIM_M * FRAGMENT_DIM_N * max_batch_size_per_block];
	__shared__ half shared_ac_f32[FRAGMENT_DIM_M * FRAGMENT_DIM_N * max_batch_size_per_block];
	__shared__ half shared_b_f16[FRAGMENT_DIM_N * FRAGMENT_DIM_N * max_batch_size_per_block];

	const auto shared_ac_fp16_ptr = shared_ac_f16 + FRAGMENT_DIM_M * FRAGMENT_DIM_N * shared_memory_id;
	const auto shared_ac_fp32_ptr = shared_ac_f32 + FRAGMENT_DIM_M * FRAGMENT_DIM_N * shared_memory_id;
	const auto shared_b_fp16_ptr = shared_b_f16 + FRAGMENT_DIM_N * FRAGMENT_DIM_N * shared_memory_id;

	// ACのコピー
	mtk::matrix_copy::g2s32x16_1w(
			shared_ac_fp16_ptr, 2 * n, n,
			ac_ptr, matrix_id * 2 * n, ac_m,
			tid
			);
	// Bのコピー
	mtk::matrix_copy::g2s16x16_1w(
			shared_b_fp16_ptr, n, n,
			b_ptr, matrix_id * n, ac_m / 2,
			tid
			);

	// TCによる行列積
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::col_major> frag_a0, frag_a1;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> frag_b;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> frag_c0, frag_c1;

	nvcuda::wmma::fill_fragment(frag_c0, 0.0f);
	nvcuda::wmma::fill_fragment(frag_c1, 0.0f);

	nvcuda::wmma::load_matrix_sync(frag_a0, shared_ac_fp16_ptr, FRAGMENT_DIM_M);
	nvcuda::wmma::load_matrix_sync(frag_a1, shared_ac_fp16_ptr + FRAGMENT_DIM_N, FRAGMENT_DIM_M);
	nvcuda::wmma::load_matrix_sync(frag_b, shared_b_fp16_ptr, FRAGMENT_DIM_N);

	nvcuda::wmma::mma_sync(frag_c0, frag_a0, frag_b, frag_c0);
	nvcuda::wmma::mma_sync(frag_c1, frag_a1, frag_b, frag_c1);

	nvcuda::wmma::store_matrix_sync(shared_ac_fp32_ptr, frag_c0, FRAGMENT_DIM_M, nvcuda::wmma::mem_col_major);
	nvcuda::wmma::store_matrix_sync(shared_ac_fp32_ptr + FRAGMENT_DIM_N, frag_c1, FRAGMENT_DIM_M, nvcuda::wmma::mem_col_major);

	mtk::matrix_copy::s2g32x16_1w(
			ac_ptr, matrix_id * 2 * n, ac_m,
			shared_ac_fp32_ptr, 2 * n, n,
			tid
			);
}

template <bool UseTC, class OUTPUT_T, class INPUT_T>
__global__ void tsqr_backward_layer0(
		OUTPUT_T* const q_ptr,
		const INPUT_T* const a_ptr,
		const INPUT_T* const b_ptr,
		const unsigned n,
		const std::size_t batch_size,
		const unsigned* const q_start_position
		) {
	constexpr std::size_t FRAGMENT_DIM_M = 32;
	constexpr std::size_t FRAGMENT_DIM_N = 16;
	constexpr std::size_t max_batch_size_per_block = 4;
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto matrix_id = tid / warp_size;
	const auto shared_memory_id = matrix_id % max_batch_size_per_block;
	const auto ac_m = q_start_position[batch_size];
	const auto q_start_pos = q_start_position[matrix_id];
	const auto sub_m = q_start_position[matrix_id + 1] - q_start_pos;

	if(matrix_id >= batch_size) return;

	__shared__ INPUT_T shared_ac_in[FRAGMENT_DIM_M * FRAGMENT_DIM_N * max_batch_size_per_block];
	__shared__ INPUT_T shared_ac_out[FRAGMENT_DIM_M * FRAGMENT_DIM_N * max_batch_size_per_block];
	__shared__ INPUT_T shared_b[FRAGMENT_DIM_N * FRAGMENT_DIM_N * max_batch_size_per_block];

	const auto shared_ac_in_ptr = shared_ac_in + FRAGMENT_DIM_M * FRAGMENT_DIM_N * shared_memory_id;
	const auto shared_ac_out_ptr = shared_ac_out + FRAGMENT_DIM_M * FRAGMENT_DIM_N * shared_memory_id;
	const auto shared_b_ptr = shared_b + FRAGMENT_DIM_N * FRAGMENT_DIM_N * shared_memory_id;

	// A(in) のコピー
	mtk::matrix_copy::g2s32x16_1w(
			shared_ac_in_ptr, sub_m, n,
			a_ptr, q_start_pos, ac_m,
			tid
			);
	// AC(out)の初期化
	mtk::matrix_operation::make_zero_matrix<INPUT_T, FRAGMENT_DIM_M, FRAGMENT_DIM_N, 1>(
			shared_ac_out_ptr, tid);
	// Bのコピー
	mtk::matrix_copy::g2s16x16_1w(
			shared_b_ptr, n, n,
			b_ptr, matrix_id * n, n * batch_size,
			tid
			);

	__syncthreads();

	mtk::gemm_core16x16<INPUT_T, 1>(
			shared_ac_out_ptr, FRAGMENT_DIM_M,
			shared_ac_in_ptr, FRAGMENT_DIM_M,
			shared_b_ptr, FRAGMENT_DIM_N,
			tid & 0x1f
			);

	mtk::gemm_core16x16<INPUT_T, 1>(
			shared_ac_out_ptr + FRAGMENT_DIM_N, FRAGMENT_DIM_M,
			shared_ac_in_ptr + FRAGMENT_DIM_N, FRAGMENT_DIM_M,
			shared_b_ptr, FRAGMENT_DIM_N,
			tid & 0x1f
			);

	__syncthreads();

	mtk::matrix_copy::s2g32x16_1w(
			q_ptr, q_start_pos, ac_m,
			shared_ac_out_ptr, sub_m, n,
			tid
			);
}

template <>
__global__ void tsqr_backward_layer0<true, float, half>(
		float* const q_ptr,
		const half* const a_ptr,
		const half* const b_ptr,
		const unsigned n,
		const std::size_t batch_size,
		const unsigned* const q_start_position
		) {
	constexpr std::size_t FRAGMENT_DIM_M = 32;
	constexpr std::size_t FRAGMENT_DIM_N = 16;
	constexpr std::size_t max_batch_size_per_block = 4;
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto matrix_id = tid / warp_size;
	const auto shared_memory_id = matrix_id % max_batch_size_per_block;
	const auto ac_m = q_start_position[batch_size];
	const auto q_start_pos = q_start_position[matrix_id];
	const auto sub_m = q_start_position[matrix_id + 1] - q_start_pos;

	if(matrix_id >= batch_size) return;

	__shared__ half shared_ac_f16[FRAGMENT_DIM_M * FRAGMENT_DIM_N * max_batch_size_per_block];
	__shared__ float shared_ac_f32[FRAGMENT_DIM_M * FRAGMENT_DIM_N * max_batch_size_per_block];
	__shared__ half shared_b_f16[FRAGMENT_DIM_N * FRAGMENT_DIM_N * max_batch_size_per_block];

	const auto shared_ac_fp16_ptr = shared_ac_f16 + FRAGMENT_DIM_M * FRAGMENT_DIM_N * shared_memory_id;
	const auto shared_ac_fp32_ptr = shared_ac_f32 + FRAGMENT_DIM_M * FRAGMENT_DIM_N * shared_memory_id;
	const auto shared_b_fp16_ptr = shared_b_f16 + FRAGMENT_DIM_N * FRAGMENT_DIM_N * shared_memory_id;

	// A のコピー
	mtk::matrix_copy::g2s32x16_1w(
			shared_ac_fp16_ptr, sub_m, n,
			a_ptr, q_start_pos, ac_m,
			tid
			);
	// Bのコピー
	mtk::matrix_copy::g2s16x16_1w(
			shared_b_fp16_ptr, n, n,
			b_ptr, matrix_id * n, n * batch_size,
			tid
			);

	// TCによる行列積
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::col_major> frag_a0, frag_a1;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> frag_b;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> frag_c0, frag_c1;

	nvcuda::wmma::fill_fragment(frag_c0, 0.0f);
	nvcuda::wmma::fill_fragment(frag_c1, 0.0f);

	nvcuda::wmma::load_matrix_sync(frag_a0, shared_ac_fp16_ptr, FRAGMENT_DIM_M);
	nvcuda::wmma::load_matrix_sync(frag_a1, shared_ac_fp16_ptr + FRAGMENT_DIM_N, FRAGMENT_DIM_M);
	nvcuda::wmma::load_matrix_sync(frag_b, shared_b_fp16_ptr, FRAGMENT_DIM_N);

	nvcuda::wmma::mma_sync(frag_c0, frag_a0, frag_b, frag_c0);
	nvcuda::wmma::mma_sync(frag_c1, frag_a1, frag_b, frag_c1);

	nvcuda::wmma::store_matrix_sync(shared_ac_fp32_ptr, frag_c0, FRAGMENT_DIM_M, nvcuda::wmma::mem_col_major);
	nvcuda::wmma::store_matrix_sync(shared_ac_fp32_ptr + FRAGMENT_DIM_N, frag_c1, FRAGMENT_DIM_M, nvcuda::wmma::mem_col_major);

	mtk::matrix_copy::s2g32x16_1w(
			q_ptr, q_start_pos, ac_m,
			shared_ac_fp32_ptr, sub_m, n,
			tid
			);
}
}

// 必要な作業用メモリ
std::size_t mtk::tsqr::get_working_q_size(const std::size_t m, const std::size_t n) {
	const auto batch_size = get_batch_size(m);
	const auto working_q_size = n * m + 2 * n * n * (batch_size - 1);

	return working_q_size;
}
std::size_t mtk::tsqr::get_working_r_size(const std::size_t m, const std::size_t n) {
	const auto batch_size = get_batch_size(m);
	const auto working_r_size_0 = n * n * batch_size;
	const auto working_r_size_1 = n * n * batch_size / 2;

	return working_r_size_0 + working_r_size_1;
}

template <bool UseTC, class T>
void mtk::tsqr::tsqr16(
		T* const q_ptr, T* const r_ptr, 
		const T* const a_ptr, const std::size_t m, const std::size_t n,
		typename get_working_q_type<T, UseTC>::type* const working_q_ptr, typename get_working_r_type<T, UseTC>::type* const working_r_ptr) {

	const std::size_t max_batch_size_per_block = 4;
	const auto batch_size_log2 = get_batch_size_log2(m);
	const auto batch_size = 1lu << batch_size_log2;
	typename get_working_r_type<T, UseTC>::type* const working_r_ptrs[2] = {working_r_ptr, working_r_ptr + n * n * batch_size};

	debug_func([&m, &n]() {std::printf("%s : matrix size = %lu x %lu\n", __func__, m, n);});
	debug_func([&batch_size]() {std::printf("%s : batch_size = %lu\n", __func__, batch_size);});
	debug_func([&working_r_ptrs]() {std::printf("%s : working_r_ptr[0] = 0x%x\n", __func__, working_r_ptrs[0]);});
	debug_func([&working_r_ptrs]() {std::printf("%s : working_r_ptr[1] = 0x%x\n", __func__, working_r_ptrs[1]);});
	debug_func([&working_q_ptr]() {std::printf("%s : working_q_ptr    = 0x%x\n", __func__, working_q_ptr);});

	const auto d_sub_m_list = cutf::memory::get_device_unique_ptr<unsigned>(batch_size + 1);
	const auto h_sub_m_list = cutf::memory::get_host_unique_ptr<unsigned>(batch_size + 1);

	// 1層目はsub_mが特殊なので別途計算を行う
	h_sub_m_list.get()[0] = 0;
	for(std::size_t i = 1; i < batch_size; i++) {
		h_sub_m_list.get()[i] = m * i / batch_size;
	}
	h_sub_m_list.get()[batch_size] = m;
	cutf::memory::copy(d_sub_m_list.get(), h_sub_m_list.get(), batch_size + 1);

	debug_func([&batch_size_log2]() {std::printf("%s : %lu bQR\n", __func__, batch_size_log2);});
	debug_func([]() {std::printf("%s : a -> wr[0]\n", __func__);});
	mtk::tcqr::qr32x16_batched<UseTC>(
			working_q_ptr,
			working_r_ptrs[0],
			a_ptr, m, n,
			batch_size, d_sub_m_list.get()
			);

	// 2層目からはsub matrixの大きさが 2n * n となるので，一度計算しGPUに転送しておけばOK
	for(std::size_t i = 0; i < batch_size / 2 + 1; i++) {
		h_sub_m_list.get()[i] = 2 * n * i;
	}
	cutf::memory::copy(d_sub_m_list.get(), h_sub_m_list.get(), batch_size / 2 + 1);

	// 再帰的QR分解のfor展開
	for(std::size_t k = batch_size_log2 - 1; k > 0; k--) {
		debug_func([&k]() {std::printf("%s : %lu bQR\n", __func__, k);});
		const auto local_batch_size = 1lu << k;	
		const auto working_q_sride = 2 * n * n * (batch_size - (1lu << (k + 1))) + m * n;
		const auto working_r_index = 1lu - (batch_size_log2 - k) % 2;
		debug_func([&working_r_index, local_batch_size]() {std::printf("%s : a(wr[%lu]) -> a(wr[%lu]) [l_bs : %lu]\n", __func__, working_r_index, 1-working_r_index, local_batch_size);});

#ifdef DEBUG_INPUT_MATRIX_PRINT
		{
			auto h_tmp = cutf::memory::get_host_unique_ptr<T>(2 * n * n * local_batch_size);
			cutf::memory::copy(h_tmp.get(), working_r_ptrs[working_r_index], 2 * n * n * local_batch_size);
			mtk::utils::print_matrix(h_tmp.get(), 2 * n * local_batch_size, n, "input");
		}
#endif

		mtk::tcqr::qr32x16_batched<UseTC>(
				working_q_ptr + working_q_sride,
				working_r_ptrs[1 - working_r_index],
				working_r_ptrs[working_r_index],
				2 * n * local_batch_size,
				n, 
				local_batch_size, d_sub_m_list.get()
				);

		debug_func([]() {CUTF_HANDLE_ERROR(hipGetLastError());});

#ifdef DEBUG_Q_MATRIX_PRINT
		{
			auto h_tmp = cutf::memory::get_host_unique_ptr<typename get_working_q_type<T, UseTC>::type>(2 * n * n * local_batch_size);
			cutf::memory::copy(h_tmp.get(), working_q_ptr + working_q_sride, 2 * n * n * local_batch_size);
			mtk::utils::print_matrix(h_tmp.get(), 2 * n * local_batch_size, n, "Q");
		}
#endif

	}

	// 最終層はrの保存先が異なる
	debug_func([]() {std::printf("%s : 1 bQR\n", __func__);});
	debug_func([&batch_size_log2]() {std::printf("%s : a(wr[%lu]) -> r\n", __func__, (batch_size_log2 % 2));});
	const auto working_q_sride = 2 * n * n * (batch_size - 2) + m * n;
	mtk::tcqr::qr32x16<UseTC>(
			working_q_ptr + working_q_sride,
			r_ptr,
			working_r_ptrs[1 - (batch_size_log2 % 2)],
			2 * n,
			n
			);

	debug_func([]() {std::printf("%s : last Q\n", __func__);});
#ifdef DEBUG_Q_MATRIX_PRINT
	{
		auto h_tmp = cutf::memory::get_host_unique_ptr<typename get_working_q_type<T, UseTC>::type>(2 * n * n);
		cutf::memory::copy(h_tmp.get(), working_q_ptr + working_q_sride, 2 * n * n);
		mtk::utils::print_matrix(h_tmp.get(), 2 * n, n, "Q");
	}
#endif

	debug_func([]() {std::printf("%s : Backword\n", __func__);});

	// Backward
	for(std::size_t k = 1; k < batch_size_log2; k++) {
		debug_func([&k]() {std::printf("%s : %lu\n", __func__, k);});
		const auto working_q_sride = 2 * n * n * (batch_size - (1lu << (k + 1))) + m * n;
		const auto grid_size = ((1lu<<k) + max_batch_size_per_block - 1) / max_batch_size_per_block;
		const auto block_size = max_batch_size_per_block * warp_size;
#ifdef DEBUG_Q_MATRIX_PRINT
		{
			const auto local_batch_size = 1lu << k;	
			auto h_tmp = cutf::memory::get_host_unique_ptr<typename get_working_q_type<T, UseTC>::type>(2 * n * n * local_batch_size);
			cutf::memory::copy(h_tmp.get(), working_q_ptr + working_q_sride, 2 * n * n * local_batch_size);
			mtk::utils::print_matrix(h_tmp.get(), 2 * n * local_batch_size, n, "Q (before backwarding)");
		}
#endif
		tsqr_backward<UseTC><<<grid_size, block_size>>>(
				working_q_ptr + working_q_sride,
				working_q_ptr + working_q_sride + (1lu << k) * 2 * n * n,
				n,
				k
				);

	}
	// 1層目はsub_mが特殊なので別途計算を行う
	h_sub_m_list.get()[0] = 0;
	for(std::size_t i = 1; i < batch_size; i++) {
		h_sub_m_list.get()[i] = m * i / batch_size;
	}
	h_sub_m_list.get()[batch_size] = m;
	cutf::memory::copy(d_sub_m_list.get(), h_sub_m_list.get(), batch_size + 1);
	const auto grid_size = (batch_size + max_batch_size_per_block - 1) / max_batch_size_per_block;
	const auto block_size = max_batch_size_per_block * warp_size;
#ifdef DEBUG_Q_MATRIX_PRINT
	{
		auto h_tmp = cutf::memory::get_host_unique_ptr<typename get_working_q_type<T, UseTC>::type>(n * m);
		cutf::memory::copy(h_tmp.get(), working_q_ptr, m * n);
		mtk::utils::print_matrix(h_tmp.get(), m, n, "Q (before backwarding)");
	}
#endif
	tsqr_backward_layer0<UseTC><<<grid_size, block_size>>>(
			q_ptr,
			working_q_ptr,
			working_q_ptr + m * n,
			n,
			batch_size,
			d_sub_m_list.get()
			);
	debug_func([]() {CUTF_HANDLE_ERROR(hipDeviceSynchronize());});
#ifdef DEBUG_Q_MATRIX_PRINT
	{
		auto h_tmp = cutf::memory::get_host_unique_ptr<T>(n * m);
		cutf::memory::copy(h_tmp.get(), q_ptr, m * n);
		mtk::utils::print_matrix(h_tmp.get(), m, n, "Q (result)");
	}
#endif
}

// (T *const q_ptr, T *const r_ptr, const T *const a_ptr, const std::size_t m, const std::size_t n, T *const working_memory_ptr)
template void mtk::tsqr::tsqr16<true, float>(float* const, float* const, const float* const, const std::size_t, const std::size_t, typename mtk::tsqr::get_working_q_type<float, true>::type* const, typename mtk::tsqr::get_working_r_type<float, true>::type* const);
template void mtk::tsqr::tsqr16<false, float>(float* const, float* const, const float* const, const std::size_t, const std::size_t, typename mtk::tsqr::get_working_q_type<float, false>::type* const, typename mtk::tsqr::get_working_r_type<float, false>::type* const);
template void mtk::tsqr::tsqr16<true, half>(half* const, half* const, const half* const, const std::size_t, const std::size_t, typename mtk::tsqr::get_working_q_type<half, false>::type* const, typename mtk::tsqr::get_working_r_type<half, false>::type* const);
template void mtk::tsqr::tsqr16<false, half>(half* const, half* const, const half* const, const std::size_t, const std::size_t, typename mtk::tsqr::get_working_q_type<half, false>::type* const, typename mtk::tsqr::get_working_r_type<half, false>::type* const);
