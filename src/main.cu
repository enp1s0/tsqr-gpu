#include <iostream>
#include <stdexcept>
#include <hip/hip_fp16.h>
#include "test.hpp"

constexpr std::size_t test_count = 16;

void qr_test(const std::vector<std::tuple<std::size_t, std::size_t, float>>& test_matrix_config_list) {
	std::cout << "# precision test" << std::endl;
	mtk::test_qr::precision<true , false, false, float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision<true , false, false, half >(test_matrix_config_list      , test_count);
	mtk::test_qr::precision<true , false, false, float, half>(test_matrix_config_list, test_count);
	mtk::test_qr::precision<false, false, false, float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision<false, false, false, half >(test_matrix_config_list      , test_count);
	mtk::test_qr::precision<true , true , false, float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision<true , false, true , float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision<true , false, true , half >(test_matrix_config_list      , test_count);
	mtk::test_qr::precision<true , false, true , float, half>(test_matrix_config_list, test_count);
	mtk::test_qr::precision<false, false, true , float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision<false, false, true , half >(test_matrix_config_list      , test_count);
	mtk::test_qr::precision<true , true , true , float>(test_matrix_config_list      , test_count);
	mtk::test_qr::cusolver_precision<float>(test_matrix_config_list                  , test_count);
	mtk::test_qr::cusolver_precision<double>(test_matrix_config_list                 , test_count);
	std::cout << "# speed test" << std::endl;
	mtk::test_qr::speed<true , false, false, float>(test_matrix_config_list      , test_count);
	mtk::test_qr::speed<true , false, false, half >(test_matrix_config_list      , test_count);
	mtk::test_qr::speed<true , false, false, float, half>(test_matrix_config_list, test_count);
	mtk::test_qr::speed<false, false, false, float>(test_matrix_config_list      , test_count);
	mtk::test_qr::speed<false, false, false, half >(test_matrix_config_list      , test_count);
	mtk::test_qr::speed<true , true , false, float>(test_matrix_config_list      , test_count);
	mtk::test_qr::speed<true , false, true , float>(test_matrix_config_list      , test_count);
	mtk::test_qr::speed<true , false, true , half >(test_matrix_config_list      , test_count);
	mtk::test_qr::speed<true , false, true , float, half>(test_matrix_config_list, test_count);
	mtk::test_qr::speed<false, false, true , float>(test_matrix_config_list      , test_count);
	mtk::test_qr::speed<false, false, true , half >(test_matrix_config_list      , test_count);
	mtk::test_qr::speed<true , true , true , float>(test_matrix_config_list      , test_count);
	mtk::test_qr::cusolver_speed<float>(test_matrix_config_list                  , test_count);
	mtk::test_qr::cusolver_speed<double>(test_matrix_config_list                 , test_count);
}


void qr_test_cond(const std::vector<std::tuple<std::size_t, std::size_t, float>>& test_matrix_config_list) {
	std::cout << "# condition number test" << std::endl;
	mtk::test_qr::precision_cond<true , true , false, float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision_cond<true , false, false, float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision_cond<false, false, false, float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision_cond<true , false, false, half >(test_matrix_config_list      , test_count);
	mtk::test_qr::precision_cond<false, false, false, half >(test_matrix_config_list      , test_count);
	mtk::test_qr::precision_cond<true , true , true , float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision_cond<true , false, true , float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision_cond<false, false, true , float>(test_matrix_config_list      , test_count);
	mtk::test_qr::precision_cond<true , false, true , half >(test_matrix_config_list      , test_count);
	mtk::test_qr::precision_cond<false, false, true , half >(test_matrix_config_list      , test_count);
	mtk::test_qr::cusolver_precision_cond<float>(test_matrix_config_list                  , test_count);
	mtk::test_qr::cusolver_precision_cond<double>(test_matrix_config_list                 , test_count);
}

int main() {
	{
		std::vector<std::tuple<std::size_t, std::size_t, float>> test_matrix_config_list;
		for (std::size_t m = 10; m <= 15; m++) {
			for (std::size_t n = 10; n <= m; n++) {
				test_matrix_config_list.push_back(std::make_tuple(1lu << m, 1lu << n, 1.0f));
			}
		}
		qr_test(test_matrix_config_list);
	}

	{
		constexpr std::size_t m = 1lu << 15;
		constexpr std::size_t n = 1lu << 7;
		std::vector<std::tuple<std::size_t, std::size_t, float>> test_matrix_config_list;
		for (std::size_t c = 2; c <= 15; c++) {
			test_matrix_config_list.push_back(std::make_tuple(m, n, 1lu << c));
		}
		qr_test_cond(test_matrix_config_list);
	}
}
