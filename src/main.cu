#include <iostream>
#include <stdexcept>
#include <hip/hip_fp16.h>
#include "test.hpp"

constexpr std::size_t min_m = 1 << 10;
constexpr std::size_t max_m = 1 << 27;
constexpr std::size_t n = 16;

int main() {
	std::cout<<"# precision test"<<std::endl;
	try{
		mtk::test::precision<true, false, float>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::precision<true, false, half>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::precision<false, false, float>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::precision<false, false, half>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::cusolver_precision<float>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::cusolver_precision<double>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};

	std::cout<<"# speed test"<<std::endl;
	try{
		mtk::test::speed<true, false, float>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::speed<true, false, half>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::speed<false, false, float>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::speed<false, false, half>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::cusolver_speed<float>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
	try{
		mtk::test::cusolver_speed<double>(min_m, max_m, n);
	}catch(std::runtime_error& e){
		std::cerr<<e.what()<<std::endl;
	};
}
