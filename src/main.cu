#include "hip/hip_runtime.h"
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>
#include <iostream>
#include <random>
#include <cmath>
#include "tcqr.hpp"
#include "utils.hpp"

int main(){
	constexpr unsigned m = 64;
	constexpr unsigned n = 16;

	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

	const std::size_t batch_size = 1u << (std::max(5u, static_cast<unsigned>( std::ceil( std::log2(static_cast<float>(m))))) - 5u);
	auto h_a_start_position = cutf::cuda::memory::get_host_unique_ptr<unsigned>(batch_size + 1);
	auto d_a_start_position = cutf::cuda::memory::get_device_unique_ptr<unsigned>(batch_size + 1);

	auto h_a = cutf::cuda::memory::get_host_unique_ptr<float>(m * n);
	auto h_q = cutf::cuda::memory::get_host_unique_ptr<float>(m * n);
	auto h_r = cutf::cuda::memory::get_host_unique_ptr<float>(n * n * batch_size);
	auto d_a = cutf::cuda::memory::get_device_unique_ptr<float>(m * n);
	auto d_q = cutf::cuda::memory::get_device_unique_ptr<float>(m * n);
	auto d_r = cutf::cuda::memory::get_device_unique_ptr<float>(n * n * batch_size);

	for(unsigned i = 0; i < batch_size; i++){
		h_a_start_position.get()[i] = i * m / batch_size;
	}
	h_a_start_position.get()[batch_size] = m;

	std::cout<<"batch size : "<<batch_size<<std::endl;
	std::cout<<"size of each batches : ";
	for(unsigned i = 0; i < batch_size; i++){
		std::cout<<(h_a_start_position.get()[i + 1] - h_a_start_position.get()[i])<<" ";
	}
	std::cout<<std::endl;
	cutf::cuda::memory::copy(d_a_start_position.get(), h_a_start_position.get(), batch_size + 1);


	for(unsigned i = 0; i < m * n; i++){
		h_a.get()[i] = dist(mt);
	}
	mtk::utils::print_matrix(
			h_a.get(), m, n, "A"
			);

	cutf::cuda::memory::copy(d_a.get(), h_a.get(), m * n);

	mtk::tcqr::qr32x16_f32tc_batched(
		d_q.get(), d_r.get(),
		d_a.get(), m, n,
		batch_size,
		d_a_start_position.get()
		);

	cutf::cuda::memory::copy(h_q.get(), d_q.get(), m * n);
	cutf::cuda::memory::copy(h_r.get(), d_r.get(), n * n * batch_size);
	hipDeviceSynchronize();
	/*mtk::utils::print_matrix(
			h_q.get(), m, n, "Q (result)"
			);
	mtk::utils::print_matrix(
			h_r.get(), n * batch_size, n, "R (result)"
			);*/
	auto d_tmp_matrix = cutf::cuda::memory::get_device_unique_ptr<float>(32 * 32);
	auto h_tmp_matrix = cutf::cuda::memory::get_host_unique_ptr<float>(32 * 32);
	auto cublas = cutf::cublas::get_cublas_unique_ptr();
	const auto one = 1.0f;
	const auto zero = 0.0f;
	for(std::size_t i = 0; i < batch_size; i++){
		const unsigned sub_m = (h_a_start_position.get()[i + 1] - h_a_start_position.get()[i]);
		cutf::cublas::gemm(
				*cublas.get(),
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				sub_m, n, n,
				&one,
				d_q.get() + h_a_start_position.get()[i], m,
				d_r.get() + i * n, batch_size * n,
				&zero,
				d_tmp_matrix.get(), sub_m
				);
		cutf::cuda::memory::copy(h_tmp_matrix.get(), d_tmp_matrix.get(), sub_m * sub_m);
		mtk::utils::print_matrix(h_q.get() + h_a_start_position.get()[i], sub_m, n, m, ("Q (" + std::to_string(i) + ")").c_str());
		mtk::utils::print_matrix(h_r.get() + i * n, n, n, batch_size * n, ("R (" + std::to_string(i) + ")").c_str());
		mtk::utils::print_matrix(h_a.get() + h_a_start_position.get()[i], sub_m, n, m, ("A (" + std::to_string(i) + ")").c_str());
		mtk::utils::print_matrix(h_tmp_matrix.get(), sub_m, n, ("QR (" + std::to_string(i) + ")").c_str());
	}
}
