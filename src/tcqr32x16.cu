#include "hip/hip_runtime.h"
#include <mma.h>
#include <hip/hip_fp16.h>
#include <cutf/type.hpp>
#include <cutf/math.hpp>
#include "matrix_copy.cuh"
#include "matrix_operations.cuh"

namespace {
constexpr unsigned warp_size = 32;

template <class Func>
__device__ void debug_func(unsigned unique_id, Func run_func){
#ifdef DEBUG
	if(unique_id == 0){
		run_func();
	}
#endif
}

template <class INPUT_T, class OUTPUT_T>
__device__ OUTPUT_T get_norm2_32(
		INPUT_T* const ptr, const unsigned size,
	   	unsigned warp_id){
	auto tmp = cutf::cuda::type::cast<OUTPUT_T>(0.0f);

	if(warp_id < size){
		tmp = cutf::cuda::type::cast<OUTPUT_T>(ptr[warp_id]);
		tmp = tmp * tmp;
	}

	for(auto mask = (warp_size >> 1); mask > 0; mask >>= 1){
		tmp += __shfl_xor_sync(0xffffffff, tmp, mask);
	}

	return cutf::cuda::type::cast<OUTPUT_T>(tmp);
}

template <class DST_T, class SRC_T, std::size_t FRAGMENT_DIM_M = 32, std::size_t FRAGMENT_DIM_N = 16>
__device__ void copy_32x16(
		DST_T* const dst_ptr,
		const SRC_T* const src_ptr,
		const unsigned unique_id
		){
	constexpr auto stride = 2 * warp_size;
	for(unsigned i = 0; i < (FRAGMENT_DIM_M * FRAGMENT_DIM_N) / stride; i++){
		dst_ptr[i * stride + unique_id] = cutf::cuda::type::cast<DST_T>(src_ptr[i * stride + unique_id]);
	}
}

template <class T, class U_T, std::size_t FRAGMENT_DIM_M = 32>
__device__ void make_h(
		T* const h_ptr, const unsigned m, 
		const U_T* const u_ptr, const U_T norm2_u_1, 
		const unsigned unique_id){
	const auto y = unique_id & 0x1f;
	const auto lane = unique_id >> 5;
	for(unsigned k = 0; k < FRAGMENT_DIM_M; k+= 2){
		const auto x = k + lane;
		U_T tmp;
		if(x == y){
			tmp = cutf::cuda::type::cast<U_T>(1.0f);
		}else{
			tmp = cutf::cuda::type::cast<U_T>(0.0f);
		}
		tmp -= cutf::cuda::type::cast<U_T>(2.0f) * u_ptr[y] * u_ptr[x] / norm2_u_1;

		h_ptr[x * FRAGMENT_DIM_M + y] = cutf::cuda::type::cast<T>(tmp);
	}
}
template <std::size_t FRAGMENT_DIM_M = 32, std::size_t FRAGMENT_DIM_N = 16>
__device__ void update_qr_f32tc(
		float* const q32_ptr, float* const r32_ptr,
		const half* const q16_ptr, const half* const r16_ptr,
		half* const h16_ptr,
		const unsigned unique_id
		){
	const auto lane = unique_id >> 5;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::col_major> h16_0_frag, h16_1_frag;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> r16_0_frag, r16_1_frag;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::col_major> q16_0_frag, q16_1_frag;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> r32_frag;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> q32_0_frag, q32_1_frag;

	nvcuda::wmma::fill_fragment(r32_frag, 0.0f);
	nvcuda::wmma::fill_fragment(q32_0_frag, 0.0f);
	nvcuda::wmma::fill_fragment(q32_1_frag, 0.0f);

	// load h
	nvcuda::wmma::load_matrix_sync(h16_0_frag, h16_ptr + FRAGMENT_DIM_N * lane, FRAGMENT_DIM_N);
	nvcuda::wmma::load_matrix_sync(h16_1_frag, h16_ptr + FRAGMENT_DIM_N * lane + FRAGMENT_DIM_M * FRAGMENT_DIM_N, FRAGMENT_DIM_N);

	/*  Q 0 */
	// load q
	nvcuda::wmma::load_matrix_sync(q16_0_frag, q16_ptr, FRAGMENT_DIM_M);
	nvcuda::wmma::load_matrix_sync(q16_1_frag, q16_ptr + FRAGMENT_DIM_N, FRAGMENT_DIM_M);
	// mma
	nvcuda::wmma::mma_sync(q32_0_frag, h16_0_frag, q16_0_frag, q32_0_frag);
	nvcuda::wmma::mma_sync(q32_0_frag, h16_1_frag, q16_1_frag, q32_0_frag);
	/*  Q 1 */
	// load q
	nvcuda::wmma::load_matrix_sync(q16_0_frag, q16_ptr + FRAGMENT_DIM_M * FRAGMENT_DIM_N, FRAGMENT_DIM_M);
	nvcuda::wmma::load_matrix_sync(q16_1_frag, q16_ptr + FRAGMENT_DIM_M * FRAGMENT_DIM_N + FRAGMENT_DIM_N, FRAGMENT_DIM_M);
	// mma
	nvcuda::wmma::mma_sync(q32_1_frag, h16_0_frag, q16_0_frag, q32_1_frag);
	nvcuda::wmma::mma_sync(q32_1_frag, h16_1_frag, q16_1_frag, q32_1_frag);
	/*  R */
	// load q
	nvcuda::wmma::load_matrix_sync(r16_0_frag, r16_ptr + FRAGMENT_DIM_M * FRAGMENT_DIM_N, FRAGMENT_DIM_M);
	nvcuda::wmma::load_matrix_sync(r16_1_frag, r16_ptr + FRAGMENT_DIM_M * FRAGMENT_DIM_N + FRAGMENT_DIM_N, FRAGMENT_DIM_M);
	// mma
	nvcuda::wmma::mma_sync(r32_frag, h16_0_frag, r16_0_frag, r32_frag);
	nvcuda::wmma::mma_sync(r32_frag, h16_1_frag, r16_1_frag, r32_frag);

	// store
	nvcuda::wmma::store_matrix_sync(q32_ptr + lane * FRAGMENT_DIM_N, q32_0_frag, FRAGMENT_DIM_M, nvcuda::wmma::mem_col_major);
	nvcuda::wmma::store_matrix_sync(q32_ptr + lane * FRAGMENT_DIM_N + FRAGMENT_DIM_M * FRAGMENT_DIM_N, q32_1_frag, FRAGMENT_DIM_M, nvcuda::wmma::mem_col_major);
	nvcuda::wmma::store_matrix_sync(r32_ptr + lane * FRAGMENT_DIM_N, r32_frag, FRAGMENT_DIM_M, nvcuda::wmma::mem_col_major);
}

template <std::size_t FRAGMENT_DIM_M = 32, std::size_t FRAGMENT_DIM_N = 16>
__device__ void qr32x16_f32tc_core(
		float* const q32_ptr, float* const r32_ptr,
		half* const q16_ptr, half* const r16_ptr,
		float* const u32_ptr, half* h16_ptr,
		const unsigned m, const unsigned n,
		const unsigned tid
		){
	const auto unique_id = tid & 0x3f;
	for(unsigned k = 0; k < n - 1; k++){
		// copy u
		// TODO ; 0埋めとデータロードを異なるwarpでできないか検証
		if(unique_id < FRAGMENT_DIM_M){
			u32_ptr[unique_id] = 0.0f;
			if(unique_id >= k){
				u32_ptr[unique_id] = r32_ptr[FRAGMENT_DIM_M * k + unique_id];
			}
		}
		__syncthreads();
		// compute |u|
		// TODO : どうせ0埋めされているなら32個で和をとってしまってもいい気がするので検証
		const auto norm_u_0 = cutf::cuda::math::sqrt<float>(get_norm2_32<float, float>(u32_ptr, m, unique_id & 0x1f));
		__syncthreads();
		// update u
		if(unique_id == k){
			u32_ptr[unique_id] += cutf::cuda::math::sign(u32_ptr[unique_id]) * norm_u_0;
		}
		__syncthreads();
		// recompute |u|
		const auto norm2_u_1 = get_norm2_32<float, float>(u32_ptr, m, unique_id & 0x1f);
		// compute h
		make_h(
				h16_ptr, m,
				u32_ptr, norm2_u_1,
				unique_id
				);
		// copy f32 to f16
		copy_32x16(r16_ptr, r32_ptr, unique_id);
		copy_32x16(q16_ptr, q32_ptr, unique_id);
		copy_32x16(q16_ptr + FRAGMENT_DIM_M * FRAGMENT_DIM_N, q32_ptr + FRAGMENT_DIM_M * FRAGMENT_DIM_N, unique_id);
		__syncthreads();
		// update q, r
		update_qr_f32tc(
				q32_ptr, r32_ptr,
				q16_ptr, r16_ptr,
				h16_ptr,
				unique_id
				);
		__syncthreads();
	}
}

template <std::size_t FRAGMENT_DIM_M = 32, std::size_t FRAGMENT_DIM_N = 16>
__global__ void qr32x16_f32_batched_kernel(
		float* const q32_ptr,
		float* const r32_ptr,
		const float* const a32_ptr,
		const unsigned m,
		const unsigned n,
		std::size_t batch_size
		){

}

template <std::size_t FRAGMENT_DIM_M = 32, std::size_t FRAGMENT_DIM_N = 16>
__global__ void qr32x16_f32_kernel(
		float* const q32_ptr,
		float* const r32_ptr,
		const float* const a32_ptr,
		const unsigned m,
		const unsigned n,
		std::size_t batch_size
		){
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float shared_q32[FRAGMENT_DIM_M * FRAGMENT_DIM_M];
	__shared__ float shared_r32[FRAGMENT_DIM_M * FRAGMENT_DIM_N];
	__shared__ half shared_q16[FRAGMENT_DIM_M * FRAGMENT_DIM_M];
	__shared__ half shared_r16[FRAGMENT_DIM_M * FRAGMENT_DIM_N];
	__shared__ half shared_h16[FRAGMENT_DIM_M * FRAGMENT_DIM_M];
	__shared__ float shared_u32[FRAGMENT_DIM_M];

	// init shared memory
	mtk::matrix_copy::g2s32x16(
			shared_r32, m, n,
			a32_ptr, 0, m,
			tid
			);
	mtk::matrix_operation::make_identity_matrix(
			shared_q32,
			tid
			);

	// qr core
	qr32x16_f32tc_core(
			shared_q32, shared_r32,
			shared_q16, shared_r16,
			shared_u32, shared_h16,
			m, n,
			tid
			);
	// store result
	mtk::matrix_copy::s2g32x16(
			q32_ptr, 0, m,
			shared_q32, m, n,
			tid
			);
	mtk::matrix_copy::s2g16x16(
			r32_ptr, 0, n,
			shared_r32, n, n,
			tid
			);
}
}
