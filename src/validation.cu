#include "hip/hip_runtime.h"
#include <cmath>
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>
#include <cutf/type.hpp>
#include "validation.hpp"
#include "matrix_copy.cuh"
#include "utils.hpp"

template <>
float mtk::validation::check_orthogonality16<float>(
		const float* const matrix,
		const std::size_t m,
		const unsigned n
		) {
	auto d_qqt = cutf::memory::get_device_unique_ptr<float>(n * n);
	auto h_qqt = cutf::memory::get_host_unique_ptr<float>(n * n);
	for(std::size_t i = 0; i < n; i++) {
		for(std::size_t j = 0; j < n; j++) {
			h_qqt.get()[i + n * j] = (i == j) ? 1.0f : 0.0f;
		}
	}
	cutf::memory::copy(d_qqt.get(), h_qqt.get(), n * n);

	auto cublas = cutf::cublas::get_cublas_unique_ptr();
	const auto alpha = 1.0f, beta = -1.0f;
	cutf::cublas::gemm(
			*cublas.get(),
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			n, n, m,
			&alpha,
			matrix, m,
			matrix, m,
			&beta,
			d_qqt.get(), n
			);
	cutf::memory::copy(h_qqt.get(), d_qqt.get(), n * n);
	float sum = 0;
	for(std::size_t i = 0; i < n * n; i++) {
		const auto tmp = h_qqt.get()[i];
		sum += tmp * tmp;
	}
	return std::sqrt(sum / n);
}

__global__ void convert_f2h(float* const dst, const half* const src, const std::size_t size){
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= size) return;

	dst[tid] = cutf::type::cast<float>(src[tid]);
}

template <>
float mtk::validation::check_orthogonality16<half>(
		const half* const matrix,
		const std::size_t m,
		const unsigned n
		) {
	constexpr std::size_t block_size = 256;
	auto d_q_f32 = cutf::memory::get_device_unique_ptr<float>(n * m);
	auto d_qqt = cutf::memory::get_device_unique_ptr<float>(n * n);
	auto h_qqt = cutf::memory::get_host_unique_ptr<float>(n * n);
	for(std::size_t i = 0; i < n; i++) {
		for(std::size_t j = 0; j < n; j++) {
			h_qqt.get()[i + n * j] = (i == j) ? 1.0f : 0.0f;
		}
	}
	cutf::memory::copy(d_qqt.get(), h_qqt.get(), n * n);
	convert_f2h<<<(m * n + block_size - 1) / block_size, block_size>>>(d_q_f32.get(), matrix, m * n);

	auto cublas = cutf::cublas::get_cublas_unique_ptr();
	const auto alpha = 1.0f, beta = -1.0f;
	cutf::cublas::gemm(
			*cublas.get(),
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			n, n, m,
			&alpha,
			d_q_f32.get(), m,
			d_q_f32.get(), m,
			&beta,
			d_qqt.get(), n
			);
	cutf::memory::copy(h_qqt.get(), d_qqt.get(), n * n);
	float sum = 0;
	for(std::size_t i = 0; i < n * n; i++) {
		const auto tmp = h_qqt.get()[i];
		sum += tmp * tmp;
	}
	return std::sqrt(sum / n);
}
