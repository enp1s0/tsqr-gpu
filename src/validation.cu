#include "hip/hip_runtime.h"
#include <cmath>
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>
#include <cutf/type.hpp>
#include "validation.hpp"
#include "matrix_copy.cuh"
#include "utils.hpp"

template<class T>
__global__ void convert_2d(double* const dst, const T* const src, const std::size_t size){
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= size) return;

	dst[tid] = cutf::type::cast<T>(src[tid]);
}

template <class T>
double mtk::validation::check_orthogonality16(
		const T* const matrix,
		const std::size_t m,
		const unsigned n
		) {
	constexpr std::size_t block_size = 256;
	auto d_q_f64 = cutf::memory::get_device_unique_ptr<double>(n * m);
	auto d_qqt = cutf::memory::get_device_unique_ptr<double>(n * n);
	auto h_qqt = cutf::memory::get_host_unique_ptr<double>(n * n);
	for(std::size_t i = 0; i < n; i++) {
		for(std::size_t j = 0; j < n; j++) {
			h_qqt.get()[i + n * j] = (i == j) ? 1.0f : 0.0f;
		}
	}
	cutf::memory::copy(d_qqt.get(), h_qqt.get(), n * n);
	convert_2d<<<(m * n + block_size - 1) / block_size, block_size>>>(d_q_f64.get(), matrix, m * n);

	auto cublas = cutf::cublas::get_cublas_unique_ptr();
	const double alpha = 1.0f, beta = -1.0f;
	cutf::cublas::gemm(
			*cublas.get(),
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			n, n, m,
			&alpha,
			d_q_f64.get(), m,
			d_q_f64.get(), m,
			&beta,
			d_qqt.get(), n
			);
	cutf::memory::copy(h_qqt.get(), d_qqt.get(), n * n);
	double sum = 0;
	for(std::size_t i = 0; i < n * n; i++) {
		const auto tmp = h_qqt.get()[i];
		sum += tmp * tmp;
	}
	return std::sqrt(sum / n);
}

template double mtk::validation::check_orthogonality16<double>(const double* const, const std::size_t, const unsigned);
template double mtk::validation::check_orthogonality16<float>(const float* const, const std::size_t, const unsigned);
template double mtk::validation::check_orthogonality16<half>(const half* const, const std::size_t, const unsigned);


template <class T>
void mtk::validation::check_submatrix_orthogonality(
		const T* const matrix,
		const std::size_t m,
		const unsigned n
		) {
	constexpr std::size_t block_size = 256;
	auto d_q_f64 = cutf::memory::get_device_unique_ptr<double>(n * m);
	auto d_qqt = cutf::memory::get_device_unique_ptr<double>(n * n);
	auto h_qqt = cutf::memory::get_host_unique_ptr<double>(n * n);
	for(std::size_t i = 0; i < n; i++) {
		for(std::size_t j = 0; j < n; j++) {
			h_qqt.get()[i + n * j] = (i == j) ? 1.0f : 0.0f;
		}
	}
	cutf::memory::copy(d_qqt.get(), h_qqt.get(), n * n);
	convert_2d<<<(m * n + block_size - 1) / block_size, block_size>>>(d_q_f64.get(), matrix, m * n);

	auto cublas = cutf::cublas::get_cublas_unique_ptr();
	const double alpha = 1.0f, beta = -1.0f;
	cutf::cublas::gemm(
			*cublas.get(),
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			n, n, m,
			&alpha,
			d_q_f64.get(), m,
			d_q_f64.get(), m,
			&beta,
			d_qqt.get(), n
			);
	cutf::memory::copy(h_qqt.get(), d_qqt.get(), n * n);

	const auto submatrix_n = 16;
	const auto submatrix_size = n / submatrix_n;
	auto submatrix_orthogonality_matrix = cutf::memory::get_host_unique_ptr<double>(submatrix_size * submatrix_size);
	for (unsigned si = 0; si < submatrix_size; si++) {
		for (unsigned sj = 0; sj < submatrix_size; sj++) {
			double orthogonality = 0;
			for (unsigned i = 0; i < submatrix_n; i++) {
				for (unsigned j = 0; j < submatrix_n; j++) {
					const auto v = h_qqt.get()[si * submatrix_n + i + (sj * submatrix_n + j) * n];
					orthogonality += v * v;
				}
			}
			submatrix_orthogonality_matrix.get()[si + sj * submatrix_size] = std::sqrt(orthogonality / submatrix_n);
		}
	}
	mtk::utils::print_matrix(submatrix_orthogonality_matrix.get(), submatrix_size, submatrix_size, submatrix_size, "sub ort matrix");
}

template void mtk::validation::check_submatrix_orthogonality<double>(const double* const, const std::size_t, const unsigned);
template void mtk::validation::check_submatrix_orthogonality<float>(const float* const, const std::size_t, const unsigned);
template void mtk::validation::check_submatrix_orthogonality<half>(const half* const, const std::size_t, const unsigned);

template <class T>
void mtk::validation::multi_orthogonality(const T* const ptr, const std::size_t ldm, const std::size_t m, const std::size_t n, const std::size_t size, hipStream_t stream) {
	hipStreamSynchronize(stream);
	auto h_mem = cutf::memory::get_host_unique_ptr<T>(m * n * size);
	cutf::memory::copy_async(h_mem.get(), ptr, m * n, stream);
	hipStreamSynchronize(stream);
	double avg_orth = 0.0;
	for (std::size_t b = 0; b < size; b++) {
		double tmp = 0.0;
		for (unsigned i = 0; i < n; i++) {
			for (unsigned j = 0; j < n; j++) {
				double c = 0.0;
				for (unsigned k = 0; k < m; k++) {
					c += cutf::type::cast<double>(h_mem.get()[i * ldm + b * m + k]) * cutf::type::cast<double>(h_mem.get()[j * ldm + b * m + k]);
				}
				double t = (c - (i == j ? 1.0 : 0.0));
				tmp += t * t;
			}
		}
		tmp = std::sqrt(tmp / n);
		std::printf("%5lu : %e\n", b, tmp);
		avg_orth += tmp;
	}
	std::printf("avg : %e\n", avg_orth / size);
}

template void mtk::validation::multi_orthogonality<half >(const half * const ptr, const std::size_t ldm, const std::size_t m, const std::size_t n, const std::size_t size, hipStream_t stream);
template void mtk::validation::multi_orthogonality<float>(const float* const ptr, const std::size_t ldm, const std::size_t m, const std::size_t n, const std::size_t size, hipStream_t stream);
