#include "hip/hip_runtime.h"
#include <iostream>
#include <matrix_copy.cuh>
#include <utils.hpp>
#include <cutf/memory.hpp>
#include <cutf/type.hpp>

constexpr std::size_t warp_size = 32;
constexpr std::size_t batch_per_block = 8;
constexpr std::size_t fragment_dim = 16;
constexpr std::size_t g_size_m = 62;
constexpr std::size_t g_size_n = fragment_dim;
constexpr std::size_t s_size_m = 12;
constexpr std::size_t s_size_n = 6;
constexpr std::size_t batch_size = (g_size_m + s_size_m -1) / s_size_m;
using test_t = float;

__global__ void kernel(const test_t* ptr){
	__shared__ float s_mem[fragment_dim * fragment_dim * batch_per_block];

	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto matrix_index = tid / warp_size;

	if(matrix_index >= batch_size) return;

	const auto s_mem_ptr = s_mem + fragment_dim * fragment_dim * matrix_index;
	const auto m = min(s_size_m, g_size_m - s_size_m * matrix_index);

	mtk::matrix_copy::g2s(
			s_mem_ptr, m, s_size_n,
			ptr, matrix_index * s_size_m, g_size_m,
			tid & 31
			);

	for(std::size_t i = 0; i < batch_size; i++){
		if(matrix_index == i && tid % 32 == 0){
			utils::print_matrix_16x16(s_mem_ptr, m, s_size_n, "mat");
		}
		__syncthreads();
	}
}

int main(){
	std::cout<<"test : "<<__FILE__<<std::endl;

	auto h_mem = cutf::cuda::memory::get_host_unique_ptr<test_t>(g_size_m * g_size_n);
	auto g_mem = cutf::cuda::memory::get_device_unique_ptr<test_t>(g_size_m * g_size_n);

	for(std::size_t i = 0; i < g_size_m * g_size_n; i++){
		h_mem.get()[i] = cutf::cuda::type::cast<test_t>(static_cast<float>(i));
	}
	utils::print_matrix(h_mem.get(), g_size_m, g_size_n, "g");
	cutf::cuda::memory::copy(g_mem.get(), h_mem.get(), g_size_m * g_size_n);

	constexpr auto grid_size = (batch_size + batch_per_block - 1) / batch_per_block;

	kernel<<<grid_size, batch_per_block * warp_size>>>(g_mem.get());

	hipDeviceSynchronize();
}
