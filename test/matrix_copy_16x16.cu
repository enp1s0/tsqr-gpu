#include "hip/hip_runtime.h"
#include <iostream>
#include <matrix_copy.cuh>
#include <utils.hpp>
#include <cutf/memory.hpp>
#include <cutf/type.hpp>

constexpr std::size_t warp_size = 32;
constexpr std::size_t batch_per_block = 8;
constexpr std::size_t fragment_dim = 16;
constexpr std::size_t g_size_m = 62;
constexpr std::size_t g_size_n = fragment_dim;
constexpr std::size_t s_size_m = 12;
constexpr std::size_t s_size_n = 6;
constexpr std::size_t batch_size = (g_size_m + s_size_m -1) / s_size_m;
using test_t = float;

__global__ void kernel16x16(test_t* ptr){
	__shared__ test_t s_mem[fragment_dim * fragment_dim * batch_per_block];

	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto matrix_index = tid / warp_size;

	if(matrix_index >= batch_size) return;

	const auto s_mem_ptr = s_mem + fragment_dim * fragment_dim * matrix_index;
	const auto m = min(s_size_m, g_size_m - s_size_m * matrix_index);

	mtk::matrix_copy::g2s16x16_1w(
			s_mem_ptr, 2, 2,
			ptr, matrix_index * s_size_m, g_size_m,
			tid
			);

	for(std::size_t i = 0; i < batch_size; i++){
		if(matrix_index == i && tid % 32 == 0){
			mtk::utils::print_matrix_16x16(s_mem_ptr, m, s_size_n, "mat");
		}
		__syncthreads();
	}

	mtk::matrix_copy::s2g16x16_1w(
			ptr, matrix_index * s_size_m, g_size_m,
			s_mem_ptr, m, s_size_n,
			tid
			);
}

int main(){
	std::cout<<"test : "<<__FILE__<<std::endl;

	auto h_mem_0 = cutf::memory::get_host_unique_ptr<test_t>(g_size_m * g_size_n);
	auto h_mem_1 = cutf::memory::get_host_unique_ptr<test_t>(g_size_m * g_size_n);
	auto g_mem = cutf::memory::get_device_unique_ptr<test_t>(g_size_m * g_size_n);

	for(std::size_t i = 0; i < g_size_m * g_size_n; i++){
		h_mem_0.get()[i] = cutf::type::cast<test_t>(static_cast<float>(i));
	}
	mtk::utils::print_matrix(h_mem_0.get(), g_size_m, g_size_n, "g");
	cutf::memory::copy(g_mem.get(), h_mem_0.get(), g_size_m * g_size_n);

	constexpr auto grid_size = (batch_size + batch_per_block - 1) / batch_per_block;

	kernel16x16<<<grid_size, batch_per_block * warp_size>>>(g_mem.get());

	cutf::memory::copy(h_mem_1.get(), g_mem.get(), g_size_m * g_size_n);
	mtk::utils::print_matrix(h_mem_1.get(), g_size_m, g_size_n, "g (g2s2g)");

	hipDeviceSynchronize();
}
