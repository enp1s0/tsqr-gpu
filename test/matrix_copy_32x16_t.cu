#include "hip/hip_runtime.h"
#include <iostream>
#include <matrix_copy.cuh>
#include <utils.hpp>
#include <cutf/memory.hpp>
#include <cutf/type.hpp>

constexpr std::size_t warp_size = 32;
constexpr std::size_t batch_per_block = 8;
constexpr std::size_t fragment_dim_m = 32;
constexpr std::size_t fragment_dim_n = 16;
constexpr std::size_t g_size_m = 64;
constexpr std::size_t g_size_n = fragment_dim_n;
constexpr std::size_t s_size_m = 32;
constexpr std::size_t s_size_n = 16;
constexpr std::size_t batch_size = (g_size_m + s_size_m - 1) / s_size_m;
using test_t = float;

__global__ void kernel32x16(test_t* const dst_ptr, const test_t* const src_ptr){
	__shared__ test_t s_mem[fragment_dim_m * fragment_dim_n * batch_per_block];

	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto matrix_index = tid / (2 * warp_size);

	if(matrix_index >= batch_size) return;

	const auto s_mem_ptr = s_mem + fragment_dim_m * fragment_dim_n * matrix_index;
	const auto m = min(s_size_m, g_size_m - s_size_m * matrix_index);

	mtk::matrix_copy::g2s32x16_2w(
			s_mem_ptr, s_size_m, s_size_n,
			src_ptr, matrix_index * s_size_m, g_size_m,
			tid
			);

	for(std::size_t i = 0; i < batch_size; i++){
		__syncthreads();
		if(matrix_index == i && tid % 64 == 0){
			mtk::utils::print_matrix_32x16(s_mem_ptr, m, s_size_n, "mat");
		}
	}

	mtk::matrix_copy::s2g32x32_16x32_t_2w(
			dst_ptr, matrix_index * 16, 32,
			s_mem_ptr, 16, 16,
			tid
			);
}

int main(){
	std::cout<<"test : "<<__FILE__<<std::endl;

	auto h_mem_0 = cutf::memory::get_host_unique_ptr<test_t>(g_size_m * g_size_n);
	auto h_mem_1 = cutf::memory::get_host_unique_ptr<test_t>(g_size_m * g_size_n);
	auto g_mem_0 = cutf::memory::get_device_unique_ptr<test_t>(g_size_m * g_size_n);
	auto g_mem_1 = cutf::memory::get_device_unique_ptr<test_t>(g_size_m * g_size_n);

	for(std::size_t i = 0; i < g_size_m * g_size_n; i++){
		h_mem_0.get()[i] = cutf::type::cast<test_t>(static_cast<float>(i));
	}
	mtk::utils::print_matrix(h_mem_0.get(), g_size_m, g_size_n, "g");
	cutf::memory::copy(g_mem_0.get(), h_mem_0.get(), g_size_m * g_size_n);

	constexpr auto grid_size = (batch_size + batch_per_block - 1) / batch_per_block;

	kernel32x16<<<grid_size, batch_per_block * warp_size * 2>>>(g_mem_1.get(), g_mem_0.get());

	cutf::memory::copy(h_mem_1.get(), g_mem_1.get(), g_size_m * g_size_n);
	mtk::utils::print_matrix(h_mem_1.get(), s_size_n * batch_size, s_size_m, "g (g2s2g)");

	hipDeviceSynchronize();
}
